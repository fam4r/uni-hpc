#include "hip/hip_runtime.h"
/* Fabrizio Margotta 789072 */

/****************************************************************************
 *
 * earthquake.c - Simple 2D earthquake model
 *
 * Copyright (C) 2018 Moreno Marzolla <moreno.marzolla(at)unibo.it>
 * Last updated on 2018-12-29
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 * ---------------------------------------------------------------------------
 *
 * Versione di riferimento del progetto di High Performance Computing
 * 2018/2019, corso di laurea in Ingegneria e Scienze Informatiche,
 * Universita' di Bologna. Per una descrizione del modello si vedano
 * le specifiche sulla pagina del corso:
 *
 * http://moreno.marzolla.name/teaching/HPC/
 *
 * Per compilare:
 *
 * gcc -D_XOPEN_SOURCE=600 -std=c99 -Wall -Wpedantic earthquake.c -o earthquake
 *
 * (il flag -D_XOPEN_SOURCE=600 e' superfluo perche' viene settato
 * nell'header "hpc.h", ma definirlo tramite la riga di comando fa si'
 * che il programma compili correttamente anche se inavvertitamente
 * non si include "hpc.h", o per errore non lo si include come primo
 * file come necessario).
 *
 * Per eseguire il programma si puo' usare la riga di comando seguente:
 *
 * ./earthquake 100000 256 > out
 *
 * Il primo parametro indica il numero di timestep, e il secondo la
 * dimensione (lato) del dominio. L'output consiste in coppie di
 * valori numerici (100000 in questo caso) il cui significato e'
 * spiegato nella specifica del progetto.
 *
 ****************************************************************************/
#include "hpc.h"
#include <stdio.h>
#include <stdlib.h>     /* rand() */
#include <assert.h>

#define BLKSIZE 1024

/* energia massima */
#define EMAX 4.0f
/* energia da aggiungere ad ogni timestep */
#define EDELTA 1e-4
/* dimensione halo */
#define HALO 1

/**
 * Restituisce un puntatore all'elemento di coordinate (i,j) del
 * dominio grid con n colonne.
 */
static inline float *IDX(float *grid, int i, int j, int n)
{
    return (grid + i*n + j);
}

/**
 * Restituisce un numero reale pseudocasuale con probabilita' uniforme
 * nell'intervallo [a, b], con a < b.
 */
float randab( float a, float b )
{
    return a + (b-a)*(rand() / (float)RAND_MAX);
}

/**
 * Inizializza il dominio grid di dimensioni n*n con valori di energia
 * scelti con probabilità uniforme nell'intervallo [fmin, fmax], con
 * fmin < fmax.
 *
 * NON PARALLELIZZARE QUESTA FUNZIONE: rand() non e' thread-safe,
 * qundi non va usata in blocchi paralleli OpenMP; inoltre la funzione
 * non si "comporta bene" con MPI (i dettagli non sono importanti, ma
 * posso spiegarli a chi e' interessato). Di conseguenza, questa
 * funzione va eseguita dalla CPU, e solo dal master (se si usa MPI).
 */
void setup( float* grid, int n, float fmin, float fmax )
{
    for ( int i=0; i<n; i++ ) {
        for ( int j=0; j<n; j++ ) {
            *IDX(grid, i, j, n) = randab(fmin, fmax);
        }
    }
}

/**
 * Somma delta a tutte le celle del dominio grid di dimensioni
 * n*n. Questa funzione realizza il passo 1 descritto nella specifica
 * del progetto.
 */
void increment_energy( float *grid, int n, float delta )
{
    for (int i=0; i<n; i++) {
        for (int j=0; j<n; j++) {
            *IDX(grid, i, j, n) += delta;
        }
    }
}

/**
 * Restituisce il numero di celle la cui energia e' strettamente
 * maggiore di EMAX.
 */
void count_cells( float *grid, int n, int *c)
{
    *c = 0;
    for (int i=0; i<n; i++) {
        for (int j=0; j<n; j++) {
            if ( *IDX(grid, i, j, n) > EMAX ) { (*c)++; }
        }
    }
}

/**
 * Distribuisce l'energia di ogni cella a quelle adiacenti (se
 * presenti). cur denota il dominio corrente, next denota il dominio
 * che conterra' il nuovo valore delle energie. Questa funzione
 * realizza il passo 2 descritto nella specifica del progetto.
 */
void propagate_energy( float *cur, float *next, int n )
{
    const float FDELTA = EMAX/4;
    for (int i=0; i<n; i++) {
        for (int j=0; j<n; j++) {
            float F = *IDX(cur, i, j, n);
            float *out = IDX(next, i, j, n);

            /* Se l'energia del vicino di sinistra (se esiste) e'
               maggiore di EMAX, allora la cella (i,j) ricevera'
               energia addizionale FDELTA = EMAX/4 */
            if ((j>0) && (*IDX(cur, i, j-1, n) > EMAX)) { F += FDELTA; }
            /* Idem per il vicino di destra */
            if ((j<n-1) && (*IDX(cur, i, j+1, n) > EMAX)) { F += FDELTA; }
            /* Idem per il vicino in alto */
            if ((i>0) && (*IDX(cur, i-1, j, n) > EMAX)) { F += FDELTA; }
            /* Idem per il vicino in basso */
            if ((i<n-1) && (*IDX(cur, i+1, j, n) > EMAX)) { F += FDELTA; }

            if (F > EMAX) {
                F -= EMAX;
            }

            /* Si noti che il valore di F potrebbe essere ancora
               maggiore di EMAX; questo non e' un problema:
               l'eventuale eccesso verra' rilasciato al termine delle
               successive iterazioni vino a riportare il valore
               dell'energia sotto la foglia EMAX. */
            *out = F;
        }
    }
}

/**
 * Restituisce l'energia media delle celle del dominio grid di
 * dimensioni n*n. Il dominio non viene modificato.
 */
void average_energy(float *grid, int n, float *Emean)
{
    float sum = 0.0f;
    for (int i=0; i<n; i++) {
        for (int j=0; j<n; j++) {
            sum += *IDX(grid, i, j, n);
        }
    }

    *Emean = (sum / (n*n));
}

int main( int argc, char* argv[] )
{
    float *cur;
    float *d_cur, *d_next;
    int s, n = 256, nsteps = 2048;
    float Emean;
    float *d_Emean;
    int c;
    int *d_c;

    srand(19); /* Inizializzazione del generatore pseudocasuale */

    if ( argc > 3 ) {
        fprintf(stderr, "Usage: %s [nsteps [n]]\n", argv[0]);
        return EXIT_FAILURE;
    }

    if ( argc > 1 ) {
        nsteps = atoi(argv[1]);
    }

    if ( argc > 2 ) {
        n = atoi(argv[2]);
    }

    /* n (e size) è la dimensione COMPRESA di HALO */
    n = n + (2 * HALO);
    const size_t domain_size = n*n*sizeof(float);
    const size_t counter_size = sizeof(int);
    const size_t emean_size = sizeof(float);

    /* Allochiamo i domini */
    cur = (float*)malloc(domain_size);
    assert(cur);

    /* Allocate space for device copies of cur, next, c*/
    CudaSafeCall(hipMalloc((void **)&d_cur, domain_size) );
    CudaSafeCall(hipMalloc((void **)&d_next, domain_size) );
    CudaSafeCall(hipMalloc((void **)&d_c, counter_size) );
    CudaSafeCall(hipMalloc((void **)&d_Emean, emean_size) );

    /* L'energia iniziale di ciascuna cella e' scelta
       con probabilita' uniforme nell'intervallo [0, EMAX*0.1] */
    setup(cur, n, 0, EMAX*0.1);

    /* Copying data from host to device */
    hipMemcpy(d_cur, &cur, domain_size, hipMemcpyHostToDevice);

    const double tstart = hpc_gettime();
    for (s=0; s<nsteps; s++) {

        /* L'ordine delle istruzioni che seguono e' importante */
        /*
        increment_energy(cur, n, EDELTA);
        c = count_cells(cur, n);
        propagate_energy(cur, next, n);
        Emean = average_energy(next, n);
        */

        increment_energy<<<1,1>>>(d_cur, n, EDELTA);

        count_cells<<<1,1>>>(d_cur, n, d_c); /* kernel must return void -> changed */
        hipMemcpy(&c, d_c, counter_size, hipMemcpyDeviceToHost);

        propagate_energy<<<1,1>>>(d_cur, d_next, n);

        average_energy<<<1,1>>>(d_next, n, d_Emean); /* kernel must return void -> changed */
        hipMemcpy(&Emean, d_Emean, emean_size, hipMemcpyDeviceToHost);

        printf("%d %f\n", c, Emean);

        /* swap cur and next on the GPU */
        float *d_tmp = d_cur;
        d_cur = d_next;
        d_next = d_tmp;
    }
    const double elapsed = hpc_gettime() - tstart;

    double Mupdates = (((double)n)*n/1.0e6)*nsteps; /* milioni di celle aggiornate per ogni secondo di wall clock time */
    fprintf(stderr, "%s : %.4f Mupdates in %.4f seconds (%f Mupd/sec)\n", argv[0], Mupdates, elapsed, Mupdates/elapsed);

    /* Libera la memoria */
    free(cur);

    /* Libera la memoria */
    hipFree(d_cur);
    hipFree(d_next);
    hipFree(d_c);
    hipFree(d_Emean);

    return EXIT_SUCCESS;
}
