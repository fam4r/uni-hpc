#include "hip/hip_runtime.h"
/* Fabrizio Margotta 789072 */

/****************************************************************************
 *
 * earthquake.c - Simple 2D earthquake model
 *
 * Copyright (C) 2018 Moreno Marzolla <moreno.marzolla(at)unibo.it>
 * Last updated on 2018-12-29
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 * ---------------------------------------------------------------------------
 *
 * Versione di riferimento del progetto di High Performance Computing
 * 2018/2019, corso di laurea in Ingegneria e Scienze Informatiche,
 * Universita' di Bologna. Per una descrizione del modello si vedano
 * le specifiche sulla pagina del corso:
 *
 * http://moreno.marzolla.name/teaching/HPC/
 *
 * Per compilare:
 *
 * gcc -D_XOPEN_SOURCE=600 -std=c99 -Wall -Wpedantic earthquake.c -o earthquake
 *
 * (il flag -D_XOPEN_SOURCE=600 e' superfluo perche' viene settato
 * nell'header "hpc.h", ma definirlo tramite la riga di comando fa si'
 * che il programma compili correttamente anche se inavvertitamente
 * non si include "hpc.h", o per errore non lo si include come primo
 * file come necessario).
 *
 * Per eseguire il programma si puo' usare la riga di comando seguente:
 *
 * ./earthquake 100000 256 > out
 *
 * Il primo parametro indica il numero di timestep, e il secondo la
 * dimensione (lato) del dominio. L'output consiste in coppie di
 * valori numerici (100000 in questo caso) il cui significato e'
 * spiegato nella specifica del progetto.
 *
 ****************************************************************************/
#include "hpc.h"
#include <stdio.h>
#include <stdlib.h>     /* rand() */
#include <assert.h>

/* We use 2D blocks of size (BLKDIM * BLKDIM) to compute
   the next configuration of the automaton */
#define BLKDIM 32

/* We use 1D blocks of (BLKDIM_REDUCTION) threads to perform reduction
 * operations */
#define BLKDIM_REDUCTION 1024

/* energia massima */
#define EMAX 4.0f
/* energia da aggiungere ad ogni timestep */
#define EDELTA 1e-4f
/* dimensione halo */
#define HALO 1

/**
 * Restituisce un puntatore all'elemento di coordinate (i,j) del
 * dominio grid con n colonne.
 * NB: n è comprensiovo di HALO.
 */
__device__ __host__ static inline float *IDX(float *grid, int i, int j, int n)
{
    return (grid + i*n + j);
}

/**
 * Restituisce un numero reale pseudocasuale con probabilita' uniforme
 * nell'intervallo [a, b], con a < b.
 */
float randab( float a, float b )
{
    return a + (b-a)*(rand() / (float)RAND_MAX);
}

/**
 * Inizializza il dominio grid di dimensioni n*n con valori di energia
 * scelti con probabilità uniforme nell'intervallo [fmin, fmax], con
 * fmin < fmax.
 *
 * NON PARALLELIZZARE QUESTA FUNZIONE: rand() non e' thread-safe,
 * qundi non va usata in blocchi paralleli OpenMP; inoltre la funzione
 * non si "comporta bene" con MPI (i dettagli non sono importanti, ma
 * posso spiegarli a chi e' interessato). Di conseguenza, questa
 * funzione va eseguita dalla CPU, e solo dal master (se si usa MPI).
 */
void setup( float* grid, int n, float fmin, float fmax )
{
    int i = 0, j = 0;

#ifdef PRINT_DEBUG
    fprintf(stderr, "setup: start internal matrix\n");
#endif
    /* Inizializzo la matrice interna (SENZA HALO) con i valori casuali */
    for(i = HALO; i < n - HALO; i++) {
        for(j = HALO; j < n - HALO; j++) {
            *IDX(grid, i, j, n) = randab(fmin, fmax);
        }
    }
#ifdef PRINT_DEBUG
    fprintf(stderr, "setup: internal matrix complete\n");
#endif

    /*
     * Note: assuming max HALO value = 1
     * If HALO would be bigger, those loops need to be handled
     * by external-looping other HALO layers (concept idea).
     */

    /* Fill matrix top and bottom with zeroes (HALO) */
    for (j = 0 ; j < n; j++) {
        *IDX(grid, 0, j, n) = 0.0f; /* TOP */
        *IDX(grid, j, 0, n) = 0.0f; /* LEFT */
        *IDX(grid, n - HALO, j, n) = 0.0f; /* BOTTOM */
        *IDX(grid, j, n - HALO, n) = 0.0f; /* RIGHT */
    }
#ifdef PRINT_DEBUG
    fprintf(stderr, "setup: halo complete\n");
#endif
}

/**
 * Somma delta a tutte le celle del dominio grid di dimensioni
 * n*n. Questa funzione realizza il passo 1 descritto nella specifica
 * del progetto.
 */
__global__ void increment_energy(float *grid, int n, float delta)
{
    const int i = HALO + blockIdx.y * blockDim.y + threadIdx.y;
    const int j = HALO + blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n - HALO && j < n - HALO) {
        *IDX(grid, i, j, n) += delta;
    }
}

/**
 * Restituisce il numero di celle la cui energia e' strettamente
 * maggiore di EMAX.
 */
__global__ void count_cells(float *grid, int n, int *c)
{
    const int i = HALO + blockIdx.x * blockDim.x + threadIdx.x;
    const int array_size = n * n;

    /* concept
       sono un thread (cella)
       se mio valore > EMAX
       scrivo 1 nella variabile
       */

    /* nelle note ho scritto che le operazioni atomiche su N grandi sono
     * dispensiose, forse non conviene lasciarlo così, provare a fare un array
     * grande n*n in cui in ogni cella un thread salva 1 o 0 in base a se il
     * valore supera EMAX e poi faccio la riduzione su quell'array */

    /* usare && */
    if (i < array_size) {
        if ( grid[i] > EMAX ) {
            atomicAdd(c, 1);
        }
    }
}

/**
 * Distribuisce l'energia di ogni cella a quelle adiacenti (se
 * presenti). cur denota il dominio corrente, next denota il dominio
 * che conterra' il nuovo valore delle energie. Questa funzione
 * realizza il passo 2 descritto nella specifica del progetto.
 */
__global__ void propagate_energy( float *cur, float *next, int n )
{
    const int i = HALO + blockIdx.y * blockDim.y + threadIdx.y;
    const int j = HALO + blockIdx.x * blockDim.x + threadIdx.x;

    const float FDELTA = EMAX/4;
    float F = *IDX(cur, i, j, n);
    float *out = IDX(next, i, j, n);

    if (i < n - HALO && j < n - HALO) {
        if ((j > 0)     && (*IDX(cur, i, j - 1, n) > EMAX)) { F += FDELTA; }
        if ((j < n - 1) && (*IDX(cur, i, j + 1, n) > EMAX)) { F += FDELTA; }
        if ((i > 0)     && (*IDX(cur, i - 1, j, n) > EMAX)) { F += FDELTA; }
        if ((i < n - 1) && (*IDX(cur, i + 1, j, n) > EMAX)) { F += FDELTA; }

        if (F > EMAX) {
            F -= EMAX;
        }

        /* Si noti che il valore di F potrebbe essere ancora
           maggiore di EMAX; questo non e' un problema:
           l'eventuale eccesso verra' rilasciato al termine delle
           successive iterazioni vino a riportare il valore
           dell'energia sotto la foglia EMAX. */
        *out = F;
    }
}

/**
 * Restituisce l'energia media delle celle del dominio grid di
 * dimensioni n*n. Il dominio non viene modificato.
 */
__global__ void average_energy(float *grid, int n, float *Emean)
{
    const int i = HALO + blockIdx.x * blockDim.x + threadIdx.x;
    const int array_size = n * n;

    if (i < array_size) {
        atomicAdd(Emean, grid[i]);
    }
}

int main( int argc, char* argv[] )
{
    float *cur;
    float *d_cur, *d_next;
    int s, width = 256, nsteps = 2048;
    float Emean;
    float *d_Emean;
    int c;
    int *d_c;

    srand(19); /* Inizializzazione del generatore pseudocasuale */

    if ( argc > 3 ) {
        fprintf(stderr, "Usage: %s [nsteps [n]]\n", argv[0]);
        return EXIT_FAILURE;
    }

    if ( argc > 1 ) {
        nsteps = atoi(argv[1]);
    }

    if ( argc > 2 ) {
        width = atoi(argv[2]);
    }

    /* width (e size) è la dimensione COMPRESA di HALO */
    width = width + (2 * HALO);
    const size_t domain_size = width*width*sizeof(float *);
    const size_t count_size = sizeof(int);
    const size_t emean_size = sizeof(float);

    /* 1D thread blocks used for reduction operations */
    dim3 reduBlock(BLKDIM_REDUCTION);
    dim3 reduGrid((width + BLKDIM_REDUCTION-1)/BLKDIM_REDUCTION);

    /* 2D thread blocks used for the update step */
    dim3 stepBlock(BLKDIM, BLKDIM);
    dim3 stepGrid((width + BLKDIM-1)/BLKDIM, (width + BLKDIM-1)/BLKDIM);

    /* Allochiamo i domini */
    cur = (float*)malloc(domain_size);
    assert(cur);

    /* Allocate space for device copies of cur, next, c*/
    cudaSafeCall(hipMalloc((void **)&d_cur, domain_size) );
    cudaSafeCall(hipMalloc((void **)&d_next, domain_size) );
    cudaSafeCall(hipMalloc((void **)&d_c, count_size) );
    cudaSafeCall(hipMalloc((void **)&d_Emean, emean_size) );
#ifdef PRINT_DEBUG
    fprintf(stderr, "hipMalloc complete\n");
#endif
    /* L'energia iniziale di ciascuna cella e' scelta
       con probabilita' uniforme nell'intervallo [0, EMAX*0.1] */
    setup(cur, width, 0, EMAX*0.1);
#ifdef PRINT_DEBUG
    fprintf(stderr, "setup complete\n");
#endif

    /* Copying data from host to device */
    hipMemcpy(&d_cur, &cur, domain_size, hipMemcpyHostToDevice);
#ifdef PRINT_DEBUG
    fprintf(stderr, "hipMemcpy (host -> device) complete\n");
#endif
    c = 0;
    hipMemcpy(d_c, &c, count_size, hipMemcpyHostToDevice);
#ifdef PRINT_DEBUG
    fprintf(stderr, "hipMemcpy (host -> device) complete\n");
#endif
    Emean = 0.0f;
    hipMemcpy(d_Emean, &Emean, emean_size, hipMemcpyHostToDevice);
#ifdef PRINT_DEBUG
    fprintf(stderr, "hipMemcpy (host -> device) complete\n");
#endif

    const double tstart = hpc_gettime();
    for (s=0; s<nsteps; s++) {
        /* L'ordine delle istruzioni che seguono e' importante */

        /* increment_energy(cur, width, EDELTA); */
        /* <<<nBlocks, nThreadsPerBlock>>> */
        increment_energy<<<stepGrid, stepBlock>>>(d_cur, width, EDELTA);
        hipDeviceSynchronize();

        /* c = count_cells(cur, width); */
        /* RIDUZIONE -> thread block 1D */
        count_cells<<<reduGrid, reduBlock>>>(d_cur, width, d_c); /* kernel must return void -> changed */
        hipDeviceSynchronize();
        hipMemcpy(&c, d_c, count_size, hipMemcpyDeviceToHost);

        /* propagate_energy(cur, next, width); */
        propagate_energy<<<stepGrid, stepBlock>>>(d_cur, d_next, width);
        hipDeviceSynchronize();

        /* Emean = average_energy(next, width); */
        /* RIDUZIONE -> thread block 1D */
        average_energy<<<reduGrid, reduBlock>>>(d_next, width, d_Emean); /* kernel must return void -> changed */
        hipDeviceSynchronize();
        hipMemcpy(&Emean, d_Emean, emean_size, hipMemcpyDeviceToHost);
        /* compute mean in CPU */
        Emean = (Emean / (width * width));

        printf("%d %f\n", c, Emean);

        /* swap cur and next on the GPU */
        float *d_tmp = d_cur;
        d_cur = d_next;
        d_next = d_tmp;
    }
    const double elapsed = hpc_gettime() - tstart;

    double Mupdates = (((double)width)*width/1.0e6)*nsteps; /* milioni di celle aggiornate per ogni secondo di wall clock time */
    fprintf(stderr, "%s : %.4f Mupdates in %.4f seconds (%f Mupd/sec)\n", argv[0], Mupdates, elapsed, Mupdates/elapsed);

    /* Free memory on host */
    free(cur);
    /* Free memory on device */
    hipFree(d_cur);
    hipFree(d_next);
    hipFree(d_c);
    hipFree(d_Emean);

    return EXIT_SUCCESS;
}
